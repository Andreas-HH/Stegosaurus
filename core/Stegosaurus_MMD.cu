#include "hip/hip_runtime.h"
#include "Stegosaurus.h"


__global__ void gammaKernel(int dim, int offset, int bw_x, int bw_y, double *down_g, double *right_g, double *results) {
  int idx_x = threadIdx.x + blockIdx.x*blockDim.x;
  int idx_y = threadIdx.y + blockIdx.y*blockDim.y;
  double temp;
  
  if (idx_x < bw_x && idx_y < bw_y) { 
    temp = down_g[idx_y*dim + offset] - right_g[idx_x*dim + offset];
    results[idx_y + bw_y*idx_x] += temp * temp;//current_sums[idx_s];
  }
}

__global__ void calcMMD(int dim, int bw_x, int bw_y, double minus_gamma, double *down_g, double *right_g, double *results, bool add) {
  int i;
  int idx_x = threadIdx.x + blockIdx.x*blockDim.x;
  int idx_y = threadIdx.y + blockIdx.y*blockDim.y;
  int idx_s = threadIdx.y + threadIdx.x*blockDim.y;
  int current_dim = blockDim.x * blockDim.y;
//   int idx_r = idx_x + idx_y*numvec; // one result per block, not per thread!
//   double current_sum = 0.;
  __shared__ double current_sums[1024]; // cuda doesn't allow me to use blockDim.x * blockDim.y here ;_;
  double temp;
  bool odd;
  
  if (idx_x < bw_x && idx_y < bw_x) { // they better all pass here, otherwise there will be a deadlock at __syncthreads()
    current_sums[idx_s] = 0.;
    // find || x - y || ^2
    for (i = 0; i < dim; i++) {
      temp = down_g[idx_y*dim + i] - right_g[idx_x*dim + i];
      current_sums[idx_s] += temp * temp;
    }
    // use gamma
    current_sums[idx_s] *= minus_gamma;
    current_sums[idx_s] = exp(current_sums[idx_s]);
    __syncthreads();
    // add the results together
    while (current_dim > 1) {
      odd = current_dim%2 == 1;
      current_dim = current_dim/2;
      if (idx_s < current_dim) {
        current_sums[idx_s] = current_sums[idx_s] + current_sums[current_dim + idx_s];
        if (odd && idx_s == 0) {
	  current_sums[idx_s] = current_sums[idx_s] + current_sums[2*current_dim];
        }
      }
      __syncthreads();
    }
  
    if (idx_s == 0) {
      if (add) results[blockIdx.y + gridDim.y * blockIdx.x] += current_sums[0];
      else     results[blockIdx.y + gridDim.y * blockIdx.x] -= 2*current_sums[0];
    }
  }
//   results[blockIdx.y + gridDim.y * blockIdx.x] = 5;
}

void initMMD(stegoContext *steg, mmdContext& mc) {
  int dim = mc.clean->dim;
  int tpb = steg->gpu_c->threads_per_block;
  
  mc.n = mc.clean->M;
  mc.kernel_blockwidth = (int) sqrt(steg->gpu_c->threads_per_block);
  mc.cache   = min(mc.n, (int) (sqrt(steg->gpu_c->doublesOnGPU / 3l + (long) SQUARE(dim) * 4l / 9l) - (long)dim * 2l / 3l));
  mc.kernel_gridwidth = (mc.cache + mc.kernel_blockwidth-1)/mc.kernel_blockwidth;
//   printf("cache: %i, kbw: %i \n", mc.cache, mc.kernel_blockwidth);
  
  CUDA_CALL( hipMalloc(&mc.clean_vectors_down_g, dim*mc.cache*sizeof(double)));
  CUDA_CALL( hipMalloc(&mc.clean_vectors_right_g, dim*mc.cache*sizeof(double)));
  CUDA_CALL( hipMalloc(&mc.stego_vectors_down_g, dim*mc.cache*sizeof(double)));
  CUDA_CALL( hipMalloc(&mc.stego_vectors_right_g, dim*mc.cache*sizeof(double)));
  CUDA_CALL( hipMalloc(&mc.results_c_vs_c_g, mc.cache*mc.cache*sizeof(double)));
  CUDA_CALL( hipMalloc(&mc.results_c_vs_s_g, mc.cache*mc.cache*sizeof(double)));
  CUDA_CALL( hipMalloc(&mc.results_s_vs_s_g, mc.cache*mc.cache*sizeof(double)));
  CUDA_CALL( hipHostAlloc(&mc.results, mc.cache*mc.cache*sizeof(double), hipHostMallocDefault));
}

void closeMMD(mmdContext& mc) {
  CUDA_CALL( hipFree(mc.clean_vectors_down_g));
  CUDA_CALL( hipFree(mc.clean_vectors_right_g));
  CUDA_CALL( hipFree(mc.stego_vectors_down_g));
  CUDA_CALL( hipFree(mc.stego_vectors_right_g));
  CUDA_CALL( hipFree(mc.results_c_vs_c_g));
  CUDA_CALL( hipFree(mc.results_c_vs_s_g));
  CUDA_CALL( hipFree(mc.results_s_vs_s_g));
  CUDA_CALL( hipHostFree(mc.results));
}

void launchGammaKernel(mmdContext& mc, int dim, int bw_x, int bw_y, double* down_g, double* right_g, double* results_g) {
  int i;
  dim3 grid, block;
  
  grid = dim3(BLOCKS(bw_x, mc.kernel_blockwidth), BLOCKS(bw_y, mc.kernel_blockwidth));    
  block = dim3(mc.kernel_blockwidth, mc.kernel_blockwidth);
  for (i = 0; i < dim; i++) {
    gammaKernel<<<grid,block>>>(dim, i, bw_x, bw_y, mc.clean_vectors_down_g, mc.clean_vectors_right_g, mc.results_c_vs_c_g);
  }
}

void estimateGamma(stegoContext *steg, mmdContext& mc) {
  int i, j;
  int bw_x, bw_y, pos_x, pos_y;
  int tpb = steg->gpu_c->threads_per_block;
  featureSet *cleanSet = mc.clean;
  int M = mc.n;
  int dim = cleanSet->dim;
  priority_queue< double > q;
  
  for (pos_x = 0l; pos_x < (long) M; pos_x += mc.cache) {
    bw_x = min(mc.cache, M-(int)pos_x);
    jumpToVector(mc.clean, pos_x);
    for (i = 0; i < bw_x; i++) {
      readVectorL2(steg, mc.clean, mc.clean_vectors_right_g + i*dim);
    }
    for (pos_y = 0l; pos_y < (long) M; pos_y += mc.cache) {
      bw_y = min(mc.cache, M-(int)pos_y);
      jumpToVector(mc.clean, pos_y);
      for (i = 0; i < bw_y; i++) {
        readVectorL2(steg, mc.clean, mc.clean_vectors_down_g + i*dim);
      }
      printf("launching kernel with parameters (%i, %i), (%i, %i), bw_x = %i, bw_y = %i \n", BLOCKS(bw_x, mc.kernel_blockwidth), BLOCKS(bw_y, mc.kernel_blockwidth), mc.kernel_blockwidth, mc.kernel_blockwidth, bw_x, bw_y);
      initDArray(mc.results_c_vs_c_g, SQUARE(mc.cache), tpb, 0.);
      launchGammaKernel(mc, dim, bw_x, bw_y, mc.clean_vectors_down_g, mc.clean_vectors_right_g, mc.results_c_vs_c_g);
      CUBLAS_CALL( hipblasGetVector(SQUARE(mc.cache), sizeof(double), mc.results_c_vs_c_g, 1, mc.results, 1));
      for (i = 0; i < bw_x; i++) {
	for (j = 0; j < bw_y; j++) {
	  if (pos_x + i == pos_y + j) continue;
	  q.push(mc.results[j + i*bw_y]);
	}
      }
    }
  }
  stegoRewind(mc.clean);
  stegoRewind(mc.stego);
  
  printf("queue size: %i, M = %i, expcted size: %i \n", q.size(), M, M*(M-1));
  for (i = 0; i < M*(M-1)/2; i++) {
    q.pop();
  }
  printf("median: %g => gamma = %g , queue size: %i \n", q.top(), 1./q.top(), q.size());
  mc.gamma = 1./q.top();
}

// probably want something block- rather than tuple-wise later!
double applyKernel(stegoContext *steg, double gamma, int dim, double *v1_g, double *v2_g, double *temp_g) {
  double norm; // use dotp instead?
  double min1 = -1;
  
  CUDA_CALL( hipMemcpy(temp_g, v2_g, dim*sizeof(double), hipMemcpyDeviceToDevice));
  CUBLAS_CALL( hipblasDaxpy(steg->gpu_c->handle, dim, &min1, v1_g, 1, temp_g, 1));
  CUBLAS_CALL( hipblasDdot(steg->gpu_c->handle, dim, temp_g, 1, temp_g, 1, &norm));
  
  return exp(-1*gamma*norm);
}

void estimateMMD(stegoContext *steg, mmdContext& mc) {
  int i, j, k;
  int bw_x, bw_y;
  int gridwidth, gridheight;
//   dim3 grid;
//   dim3 block;
  long pos_x, pos_y;
  int tpb = steg->gpu_c->threads_per_block;
//   int cache = mc->cache;
//   int blockwidth_g, pos_g, cache_g = mc->cache_g;
//   int lessThanOne = 0, greaterThanOne = 0;
//   double *vectors;
  int dim = mc.clean->dim;
  int M = mc.n;
  double mmd = 0.;
  double temp = 0.;
  
  for (pos_x = 0l; pos_x < (long) M; pos_x += mc.cache) {
    bw_x = min(mc.cache, M-(int)pos_x);
//     printf("clean [%i] \n", pos_x);
    jumpToVector(mc.clean, pos_x);
//     printf("stego [%i] \n", pos_x);
    jumpToVector(mc.stego, pos_x);
    for (i = 0; i < bw_x; i++) {
      readVectorRescaled(steg, mc.clean, mc.clean_vectors_down_g + i*dim);
    } // better to one after the other, might be quicker
    for (i = 0; i < bw_x; i++) {
      readVectorRescaled(steg, mc.stego, mc.stego_vectors_down_g + i*dim);
    }
    for (pos_y = 0l; pos_y < (long) M; pos_y += mc.cache) {
      bw_y = min(mc.cache, M-(int)pos_y);
//       printf(" clean [%i] \n", pos_y);
      jumpToVector(mc.clean, pos_y);
//       printf(" stego [%i] \n", pos_y);
      jumpToVector(mc.stego, pos_y);
      for (i = 0; i < bw_x; i++) {
        readVectorRescaled(steg, mc.clean, mc.clean_vectors_right_g + i*dim);
      }
      for (i = 0; i < bw_x; i++) {
        readVectorRescaled(steg, mc.stego, mc.stego_vectors_right_g + i*dim);
      }
//       for (j = 0; j < bw_x; j++) {
// 	for (k = 0; k < bw_y; k++) {
// 	  if (pos_x+j == pos_y + k)  continue;
// 	  mmd += applyKernel(steg, mc.gamma, dim, mc.clean_vectors_down_g+j*dim, mc.clean_vectors_right_g+k*dim, mc.temp_g);
// 	  mmd += applyKernel(steg, mc.gamma, dim, mc.stego_vectors_down_g+j*dim, mc.stego_vectors_right_g+k*dim, mc.temp_g);
// 	  mmd -= 2.*applyKernel(steg, mc.gamma, dim, mc.clean_vectors_down_g+j*dim, mc.stego_vectors_right_g+k*dim, mc.temp_g);
// 	}
//       }
      gridwidth = (bw_x + mc.kernel_blockwidth-1)/mc.kernel_blockwidth;
      gridheight = (bw_y + mc.kernel_blockwidth-1)/mc.kernel_blockwidth;
      initDArray(mc.results_c_vs_c_g, mc.kernel_gridwidth*mc.kernel_gridwidth, tpb, 0.);
      calcMMD<<<(gridwidth, gridheight), (mc.kernel_blockwidth, mc.kernel_blockwidth)>>>(dim, bw_x, bw_y, -1.*0.2, mc.clean_vectors_down_g, mc.clean_vectors_right_g, mc.results_c_vs_c_g, true);
      calcMMD<<<(gridwidth, gridheight), (mc.kernel_blockwidth, mc.kernel_blockwidth)>>>(dim, bw_x, bw_y, -1.*0.2, mc.stego_vectors_down_g, mc.stego_vectors_right_g, mc.results_c_vs_c_g, true);
      calcMMD<<<(gridwidth, gridheight), (mc.kernel_blockwidth, mc.kernel_blockwidth)>>>(dim, bw_x, bw_y, -1.*0.2, mc.clean_vectors_down_g, mc.stego_vectors_right_g, mc.results_c_vs_c_g, false);
//       CUBLAS_CALL( hipblasGetMatrix(gridheight, gridwidth, sizeof(double), mc.results_c_vs_c_g, mc.kernel_gridwidth, mc.results, mc.kernel_gridwidth));
      CUBLAS_CALL( hipblasGetVector(SQUARE(mc.kernel_gridwidth), sizeof(double), mc.results_c_vs_c_g, 1, mc.results, 1));
      temp = 0.;
      printf("results[10]: %g \n", mc.results[10]);
      for (j = 0; j < gridwidth; j++) {
	for (k = 0; k < gridheight; k++) {
	  if (pos_x+j == pos_y + k)  continue;
	  temp += mc.results[j*mc.kernel_gridwidth + k];
//           printf("%g ", mc.results[j*gridwidth + k]);
	  mmd += mc.results[j*gridwidth + k];
	}
      }
      printf("delta mmd: %f \n", temp);
    }
//     break;
//     stegoRewind(mc.stego);
  }
  stegoRewind(mc.clean);
  stegoRewind(mc.stego);
  printf("have some mmd: %f \n", mmd);
  mc.mmd = 1./((double) M*(M-1)) * mmd;
}